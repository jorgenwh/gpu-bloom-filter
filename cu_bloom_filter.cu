#include <inttypes.h>
#include <string.h>

#include "cu_bloom_filter.h"
#include "kernels.h"

CuBloomFilter::CuBloomFilter(const int size) {
  size_m = size;
  hipMalloc(&table_m, sizeof(uint64_t)*size);
  hipMemset(table_m, 0, sizeof(uint64_t)*size);
}

void CuBloomFilter::insert(const uint64_t *values, const int size) {
  uint64_t *d_values;
  hipMalloc(&d_values, sizeof(uint64_t)*size);
  hipMemcpy(d_values, values, sizeof(uint64_t)*size, hipMemcpyHostToDevice);
  insert_cu_bloom_filter(d_values, size, table_m, size_m);
  hipFree(d_values);
}

uint64_t CuBloomFilter::get(const uint64_t key) const {
  uint64_t value;
  uint64_t hash = key % size_m;
  hipMemcpy(&value, &table_m[hash], sizeof(uint64_t), hipMemcpyDeviceToHost);
  return value;
}
