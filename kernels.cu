#include "hip/hip_runtime.h"
#include <inttypes.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "kernels.h"

__global__ void insert_cu_bloom_filter_kernel(
    const uint64_t *keys, const int keyset_size, uint64_t *table, int table_size) 
{
  int key_idx = blockIdx.x * blockDim.x + threadIdx.x;
  uint64_t hash = keys[key_idx] % table_size;

  atomicAdd_system(reinterpret_cast<unsigned long long *>(&table[hash]), 1);
}

void insert_cu_bloom_filter(
    const uint64_t *keys, const int keyset_size, uint64_t *table, int table_size) 
{
  uint32_t num_blocks = keyset_size / THREADS_PER_BLOCK + (keyset_size % THREADS_PER_BLOCK > 0);
  insert_cu_bloom_filter_kernel<<<num_blocks, THREADS_PER_BLOCK>>>(
      keys, keyset_size, table, table_size);
  hipDeviceSynchronize();
}
